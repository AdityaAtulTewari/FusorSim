#include "hip/hip_runtime.h"
#include <sim.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <ofstream>

constexpr GOOD_BLOCK_SIZE = 64;

__device__
static real_v force_part(const double& q0[REAL_V_DIM], const double& q1[REAL_V_DIM], double c0, double c1)
{
  double diff[REAL_V_DIM];
  double dist = 0.0;
  for(int i = 0; i < REAL_V_DIM; i++)
  {
    diff[i] = q1[i] - q0[i];
    double dist += diff[i] * diff[i];
  }
  //Deal with edge issues
  if(dist == 0.0)
  {
    dist = 1e-300;
    for(int i = 0; i < REAL_V_DIM; i++) diff[i] = 1e-10;
  }
  dist = sqrt(dist);
  double mag_F = -c0 * c1 /dist;
  real_v ret;
  for(int i = 0; i < REAL_V_DIM; i++) ret.vals[i] = diff[i] * mag_F/dist;
  return ret;
}

__global__
static void compute_forces(real_v* q0, real_v* c, real_v* out, uint64 N)
{
  int id = blockIdx.x * blockDim.x + blockIdx.x;
  //out is an array of force components, for each particle there are N forces, for N particles
  int big = floor((sqrt(8*id +1)-1)/2);
  int lil = id - y*(y+1)/2;
  int q0i = big;
  int q1i = lil;
  if(q1i == q0i) q1i = N;
  real_v to_out = force_part(q0[q0i].val, q0[q1i].val, c[q0i], c[q1i]);
  out[big*N + lil] = to_out;
  if(q1i < N)
  {
    for(int i = 0; i < REAL_V_DIM; i++) out[lil*N + big].vals[i] = -to_out.vals[i];
  }
}

__global__
static void q1_update(real_v* q0, real_v* qdot0, double dt, uint64 N)
{
  id = blockIdx.x * blockDim.x + blockIdx.x;
  if(N < dim)
  {
    q0[id].vals[blockIdx.y] += dt * qdot0[id].vals[blockIdx.y];
  }
}

__global__
static void qdot1_update(real_v* qdot0, real_v* Fq1, double dt, uint64 N, double* m)
{
  id = blockIdx.x * blockDim.x + blockIdx.x;
  if(N < dim)
  {
    q0[id].vals[blockIdx.y] += dt * Fq1[id].vals[blockIdx.y]/m[dim];
  }
}

static struct comp_Fq1
{
  comp_Fq1() {}
  real_v operator() (real_v a, real_v b)
  {
    real_v ret;
    for(int i = 0; i < REAL_V_DIM; i++) ret[i] = a[i] + b[i];
    return ret;
  }
};

thrust::equal_to<int> pred;
comp_Fq1 spec_add;

hipStream_t stream1, stream2;

constexpr dim3 q1_thread(GOOD_BLOCK_SIZE, REAL_V_DIM);
constexpr int forces_thread = GOOD_BLOCK_SIZE;
constexpr int streams = 6;

static void step(real_v* q, real_v* qdot, double* m, double* c, double dt, uint64 N, real_v* out, real_v* Fq1, real_v* temp_q, const hipStream_t& s[streams], void (*passed)(output), output prepped)
{
  hipEvent_t event;
  hipEventCreate(&event);
  int q1_blocks = (N % GOOD_BLOCK_SIZE) ? (N/GOOD_BLOCK_SIZE) + 1 : N/GOOD_BLOCK_SIZE;
  //Update q1
  q1_update<<<q1_blocks, q1_thread,0,s1>>>(q, qdot, dt, N);
  hipEventRecord(event, s[0]);
  hipStreamWaitEvent(s[1], event);
  hipMemcpyAsync(temp_q, q, sizeof(real_v)*N, hipMemcpyDeviceToHost, s[1]);
  //Update qdot1
  //Compute forces
  int blah = N*(N+1) /2;
  int forces_blocks = (blah % GOOD_BLOCK_SIZE) ? (blah / GOOD_BLOCK_SIZE) + 1 : N / GOOD_BLOCK_SIZE;
  compute_forces<<<forces_blocks, forces_thread,0,s[0]>>>(q, c, out, N);
  auto counter = thrust:make_counting_iterator(0);
  auto philled = repeat_iterator<thrust::counting_iterator<int>>(counter, N);
  thrust::reduce_by_key(thrust::cuda::par.on(s[0]), philled, philled + N * N, out, thrust::make_discard_iterator(), Fq1, pred, spec_add);
  qdot1_update<<<q1_blocks, q1_thread,0,s[0]>>>(qdot, Fq1, dt, N, m);
  hipStreamSynchronize(s[1]);
  prepped.part = temp_q;
  passed(prepped);
  hipDeviceSynchronize();
}

output* n_body_eval(real_v* q, real_v* v, double* m, double* c, double QE, double r, double dt, uint64 n, uint64 N, void (*passed)(output))
{
  real_v* dev_q;
  real_v* dev_v;
  double* dev_m;
  double* dev_c;
  real_v* d_out;
  real_v* d_Fq1;
  hipStream_t s[streams];
  for(int i = 0; i < streams; i++) hipStreamCreate(&s[i]);
  output prep;
  //For optimization place the central charge in the same array
  hipMalloc(dev_q, sizeof(real_v)*(N+1));
  hipMalloc(dev_v, sizeof(real_v)*N);
  hipMalloc(dev_m, sizeof(double)*N);
  hipMalloc(dev_c, sizeof(double)*(N+1));
  hipMalloc(d_out, sizeof(real_v)*N*N);
  hipMalloc(d_Fq1, sizeof(real_v)*N);
  hipDeviceSynchronize();
  hipMemcpyAsync(dev_q, q, sizeof(real_v)*N, hipMemcpyHostToDevice, s[0]);
  hipMemcpyAsync(dev_v, v, sizeof(real_v)*N, hipMemcpyHostToDevice, s[1]);
  hipMemcpyAsync(dev_m, m, sizeof(double)*N, hipMemcpyHostToDevice, s[2]);
  hipMemcpyAsync(dev_c, c, sizeof(double)*N, hipMemcpyHostToDevice, s[3]);
  hipMemcpyAsync(&dev_c[N],&QE,sizeof(double),hipMemcpyHostToDevice,s[4]);
  hipMemsetAsync(&dev_q[N],0,sizeof(real_v), hipMemcpyHostToDevice, s[5]);
  hipDeviceSynchronize();
  out.numb = N;
  out.part = nullptr;
  for(uint64 i = 0; i < n; i++)
  {
    out.step = i;
    step(dev_q, dev_v, dev_m, dev_c, dt, N, d_out, d_Fq1, q, s, passed);
  }
  hipFree(dev_q);
  hipFree(dev_v);
  hipFree(dev_m);
  hipFree(dev_c);
  hipFree(d_out);
  hipFree(d_Fq1);
  hipDeviceSynchronize();

}
